#include "hip/hip_runtime.h"
//
// Created by giost on 05/02/2025.
//

#include "histogram_equalization_cuda.h"
#include <hip/hip_runtime.h>
#include <>

using namespace cv;

// Kernel 1 per calcolare l'istogramma con memoria condivisa
__global__ void computeHistogram(const uchar* input, int* hist, int width, int height) {
    __shared__ int local_hist[256]; // Memoria condivisa:si usa memoria shared per accumulare un istogramma locale (migliora le performance, riducendo il traffico con la memoria globale)
    int tid = threadIdx.x + threadIdx.y * blockDim.x; // Serve per indicizzare l'array dell'istogramma.

    // Inizializza l'istogramma locale
    if (tid < 256) {
        local_hist[tid] = 0;
    }
    __syncthreads();

    // Calcola le coordinate globali
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Aggiorna l'istogramma locale
    if (x < width && y < height) {
        int pixel_value = input[y * width + x]; // Accesso coalescente
        atomicAdd(&local_hist[pixel_value], 1);
    }
    __syncthreads();

    // Unisci gli istogrammi locali in quello globale
    if (tid < 256) {
        atomicAdd(&hist[tid], local_hist[tid]);
    }
}

// Kernel 2 per calcolare la CDF
// Si usa un Parallel Prefix Sum (Scan) per velocizzare il calcolo della CDF
__global__ void computeCDF(int* hist, int* cdf) {
    __shared__ int temp[256]; // Memoria condivisa

    int tid = threadIdx.x;

    // Carica l'istogramma nella memoria condivisa
    temp[tid] = hist[tid];
    __syncthreads();

    // **Up-sweep (Riduzione)**
    for (int offset = 1; offset < 256; offset *= 2) {
        int val = 0;
        if (tid >= offset) {
            val = temp[tid - offset];
        }
        __syncthreads();
        temp[tid] += val;
        __syncthreads();
    }

    // Scrivi il risultato finale
    cdf[tid] = temp[tid];
}

// Kernel 3 per applicare la trasformazione
__global__ void applyTransformation(uchar* output, const uchar* input, const uchar* lookup_table, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        output[idx] = lookup_table[input[idx]]; // Accesso coalescente
    }
}

void histogram_equalization_cuda(const Mat& input, Mat& output) {
    int width = input.cols; // Larghezza sarà quella della immagine che viene passata in input (colonne delle img)
    int height = input.rows; // Altezza sarà quella della immagine che viene passata in input (righe delle img)
    int total_pixels = width * height;

    // Alloca memoria sulla GPU (device) (NB: h = host, d = device)
    uchar* d_input;
    uchar* d_output;
    int* d_hist; // Istogramma
    int* d_cdf; // CDF
    uchar* d_lookup_table; // Tabella di lookup
    // In questo modo si ha che dati di ogni attributo sono separati in array distinti + efficace per parallelismo su GPU xché permette accesso coalescente a memoria
    // --> Ogni struttura dati è rappresentata come un array indipendente.

    hipMalloc((void**)&d_input, total_pixels * sizeof(uchar));
    hipMalloc((void**)&d_output, total_pixels * sizeof(uchar));
    hipMalloc((void**)&d_hist, 256 * sizeof(int));
    hipMalloc((void**)&d_cdf, 256 * sizeof(int));
    hipMalloc((void**)&d_lookup_table, 256 * sizeof(uchar));

    // Usa Pinned Memory per le strutture sulla CPU (host) (NB: h = host, d = device) --> in modo da velocizzare i trasferimenti da GPU a CPU
    int* h_cdf;
    uchar* h_lookup_table;
    hipHostAlloc((void**)&h_cdf, 256 * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_lookup_table, 256 * sizeof(uchar), hipHostMallocDefault);

    // Copia l'immagine input sulla GPU
    hipMemcpy(d_input, input.data, total_pixels * sizeof(uchar), hipMemcpyHostToDevice);

    // Inizializza l'istogramma a zero
    hipMemset(d_hist, 0, 256 * sizeof(int));

    // Definizione eventi CUDA per il timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Parametri per i kernel
    dim3 blockSize(32, 16); // Dimensione del blocco è data da 32x16 (512 threads) // Provando a cambiare con 32x32 non cambia molto
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // Inizia la misurazione del tempo solo per i kernel
    hipEventRecord(start);

    // Kernel 1: Calcolo dell'istogramma
    computeHistogram<<<gridSize, blockSize>>>(d_input, d_hist, width, height);
    //hipDeviceSynchronize(); // Sincronizza prima di passare alla CDF --> rimossa perché non necessaria

    // Kernel 2: Calcolo della CDF
    computeCDF<<<1, 256>>>(d_hist, d_cdf);
    //hipDeviceSynchronize(); // Sincronizza prima di passare alla applyTransformation --> rimossa perché non necessaria

    // Copia la CDF dalla GPU alla CPU (più veloce grazie alla Pinned Memory) [senza sincronizzazione]
    hipMemcpyAsync(h_cdf, d_cdf, 256 * sizeof(int), hipMemcpyDeviceToHost);

    // Aspetta che la CDF sia copiata sulla CPU prima di procedere con la lookup table
    hipDeviceSynchronize(); // Questa sincronizzazione è necessaria per evitare che la lookup table venga calcolata prima che h_cdf sia pronto

    // Calcolo della lookup table sulla CPU
    int min_cdf = h_cdf[0];
    for (int i = 1; i < 256; i++) {
        if (h_cdf[i] < min_cdf) {
            min_cdf = h_cdf[i];
        }
    }

    for (int i = 0; i < 256; i++) {
        float value = ((h_cdf[i] - min_cdf) * 255.0f) / (total_pixels - min_cdf);
        h_lookup_table[i] = static_cast<uchar>(std::min(std::max(value, 0.0f), 255.0f));
    }

    // Copia la lookup table sulla GPU (più veloce grazie alla Pinned Memory)
    hipMemcpyAsync(d_lookup_table, h_lookup_table, 256 * sizeof(uchar), hipMemcpyHostToDevice);

    // Kernel 3: Applicazione della trasformazione (applica la tabella di lookup all'immagine)
    applyTransformation<<<gridSize, blockSize>>>(d_output, d_input, d_lookup_table, width, height);

    // Sincronizza solo prima di operazioni critiche, non per ogni kernel
    hipDeviceSynchronize();

    // Registra il tempo di fine
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calcola il tempo impiegato dai kernel
    float kernel_time;
    // Si prende il tempo passato tra i due eventi start e stop
    hipEventElapsedTime(&kernel_time, start, stop); // Serve calcolare tempo totale esecuzione dei 3 kernel senza considerare copie di memoria (hipMemcpy) --> prende il tmepo tra i due eventi (start e stop)
    std::cout << "Tempo di esecuzione solo dei kernel CUDA: " << kernel_time << " ms" << std::endl;

    // Copia il risultato sulla CPU (più veloce grazie alla Pinned Memory)
    hipMemcpy(output.data, d_output, total_pixels * sizeof(uchar), hipMemcpyDeviceToHost); // Questa operazione è quella che rallenta l'esecuzione

    // Libera memoria GPU
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_hist);
    hipFree(d_cdf);
    hipFree(d_lookup_table);

    // Libera la memoria pinned sulla CPU
    hipHostFree(h_cdf);
    hipHostFree(h_lookup_table);

    // Distrugge gli eventi CUDA
    hipEventDestroy(start);
    hipEventDestroy(stop);
}



