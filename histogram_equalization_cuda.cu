#include "hip/hip_runtime.h"
//
// Created by giost on 05/02/2025.
//

#include "histogram_equalization_cuda.h"
#include <hip/hip_runtime.h>
#include <>

using namespace cv;

// Kernel 1 per calcolare l'istogramma con memoria condivisa
__global__ void computeHistogram(const uchar* input, int* hist, int width, int height) {
    __shared__ int local_hist[256];  // Istogramma locale in memoria shared // Memoria condivisa:si usa memoria shared per accumulare un istogramma locale (migliora le performance, riducendo il traffico con la memoria globale)
    __shared__ uchar tile[16][16];   // Memoria shared per un tile 16x16

    int tid = threadIdx.x + threadIdx.y * blockDim.x; // Indice lineare del thread // Serve per indicizzare l'array dell'istogramma.

    // Inizializza l'istogramma locale
    if (tid < 256) {
        local_hist[tid] = 0;
    }
    __syncthreads();

    // Calcola le coordinate globali e locali
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Carichiamo i pixel nella shared memory per il Tile (Utilizzo della memoria shared per i pixel dell'immagine (tile[32][32]))
    if (x < width && y < height) {
        tile[threadIdx.y][threadIdx.x] = input[y * width + x];
    }
    __syncthreads(); // All threads in the same block must reach the __syncthreads() before any of the them can move on

    // Ogni thread aggiorna l'istogramma locale usando la memoria shared (Calcolo dell'istogramma direttamente sulla shared memory)
    if (x < width && y < height) {
        int pixel_value = tile[threadIdx.y][threadIdx.x];
        atomicAdd(&local_hist[pixel_value], 1);
    }
    __syncthreads(); // All threads in the same block must reach the __syncthreads() before any of the them can move on

    // Uniamo i risultati dell'istogramma locale con la memoria globale (Scrittura ottimizzata della memoria globale)
    if (tid < 256) {
        atomicAdd(&hist[tid], local_hist[tid]);
    }
}

// Kernel 2 per calcolare la CDF
// Si usa un Parallel Prefix Sum (Scan) per velocizzare il calcolo della CDF
__global__ void computeCDF(int* hist, int* cdf) {
    __shared__ int temp[256]; // Memoria condivisa

    int tid = threadIdx.x;

    // Carica l'istogramma nella memoria condivisa
    temp[tid] = hist[tid];
    __syncthreads();

    // **Up-sweep (Riduzione)**
    for (int offset = 1; offset < 256; offset *= 2) {
        int val = 0;
        if (tid >= offset) {
            val = temp[tid - offset];
        }
        __syncthreads();
        temp[tid] += val;
        __syncthreads();
    }

    // Scrivi il risultato finale
    cdf[tid] = temp[tid];
}

// Kernel 3 per applicare la trasformazione
__global__ void applyTransformation(uchar* output, const uchar* input, const uchar* lookup_table, int width, int height) {
    __shared__ uchar tile[16][16]; // Memoria condivisa per un Tile 16x16

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Carichiamo un Tile nella shared memory
    if (x < width && y < height) {
        tile[threadIdx.y][threadIdx.x] = input[y * width + x];
    }
    __syncthreads(); // All threads in the same block must reach the __syncthreads() before any of the them can move on

    // Applicazione della trasformazione
    if (x < width && y < height) {
        output[y * width + x] = lookup_table[tile[threadIdx.y][threadIdx.x]]; // Accesso Coalescente
    }
}

void histogram_equalization_cuda(const Mat& input, Mat& output) {
    int width = input.cols; // Larghezza sarà quella della immagine che viene passata in input (colonne delle img)
    int height = input.rows; // Altezza sarà quella della immagine che viene passata in input (righe delle img)
    int total_pixels = width * height;

    // Informazioni utili per il calcolo dell'Occupancy (Kernel 1 computeHistogram)
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))computeHistogram);
    std::cout << "Registri per thread (kernel 1): " << attr.numRegs << std::endl;
    std::cout << "Shared memory per blocco (kernel 1): " << attr.sharedSizeBytes << " bytes" << std::endl;

    // Informazioni utili per il calcolo dell'Occupancy (Kernel 2 computeCDF)
    hipFuncAttributes attr2;
    hipFuncGetAttributes(&attr2, reinterpret_cast<const void*>((const void*))computeCDF);
    std::cout << "Registri per thread (kernel 2): " << attr2.numRegs << std::endl;
    std::cout << "Shared memory per blocco (kernel 2): " << attr2.sharedSizeBytes << " bytes" << std::endl;

    // Informazioni utili per il calcolo dell'Occupancy (Kernel 3 applyTransformation)
    hipFuncAttributes attr3;
    hipFuncGetAttributes(&attr3, reinterpret_cast<const void*>((const void*))applyTransformation);
    std::cout << "Registri per thread (kernel 3): " << attr3.numRegs << std::endl;
    std::cout << "Shared memory per blocco (kernel 3): " << attr3.sharedSizeBytes << " bytes" << std::endl;

    // Alloca memoria sulla GPU (device) (NB: h = host, d = device)
    uchar* d_input;
    uchar* d_output;
    int* d_hist; // Istogramma
    int* d_cdf; // CDF
    uchar* d_lookup_table; // Tabella di lookup
    // In questo modo si ha che dati di ogni attributo sono separati in array distinti + efficace per parallelismo su GPU xché permette accesso coalescente a memoria
    // --> Ogni struttura dati è rappresentata come un array indipendente.

    hipMalloc((void**)&d_input, total_pixels * sizeof(uchar));
    hipMalloc((void**)&d_output, total_pixels * sizeof(uchar));
    hipMalloc((void**)&d_hist, 256 * sizeof(int));
    hipMalloc((void**)&d_cdf, 256 * sizeof(int));
    hipMalloc((void**)&d_lookup_table, 256 * sizeof(uchar));

    // Usa Pinned Memory per le strutture sulla CPU (host) (NB: h = host, d = device) --> in modo da velocizzare i trasferimenti da GPU a CPU
    int* h_cdf;
    uchar* h_lookup_table;
    hipHostAlloc((void**)&h_cdf, 256 * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_lookup_table, 256 * sizeof(uchar), hipHostMallocDefault);

    // Copia l'immagine input sulla GPU
    hipMemcpy(d_input, input.data, total_pixels * sizeof(uchar), hipMemcpyHostToDevice);

    // Inizializza l'istogramma a zero
    hipMemset(d_hist, 0, 256 * sizeof(int));

    // Definizione eventi CUDA per il timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Parametri per i kernel
    dim3 blockSize(16, 16); // Dimensione del blocco è data da 16x16 (256 threads) // Provando a cambiare con 32x32 non cambia molto
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // Inizia la misurazione del tempo solo per i kernel
    hipEventRecord(start);

    // Kernel 1: Calcolo dell'istogramma
    computeHistogram<<<gridSize, blockSize>>>(d_input, d_hist, width, height);
    //hipDeviceSynchronize(); // Sincronizza prima di passare alla CDF --> rimossa perché non necessaria (rallenta esecuzione)

    // Kernel 2: Calcolo della CDF
    computeCDF<<<1, 256>>>(d_hist, d_cdf);
    //hipDeviceSynchronize(); // Sincronizza prima di passare alla applyTransformation --> rimossa perché non necessaria (rallenta esecuzione)

    // Copia la CDF dalla GPU alla CPU (più veloce grazie alla Pinned Memory) [senza sincronizzazione]
    hipMemcpyAsync(h_cdf, d_cdf, 256 * sizeof(int), hipMemcpyDeviceToHost);

    // Aspetta che la CDF sia copiata sulla CPU prima di procedere con la lookup table
    hipDeviceSynchronize(); // Questa sincronizzazione è necessaria per evitare che la lookup table venga calcolata prima che h_cdf sia pronto

    // Calcolo della lookup table sulla CPU
    int min_cdf = h_cdf[0];
    for (int i = 1; i < 256; i++) {
        if (h_cdf[i] < min_cdf) {
            min_cdf = h_cdf[i];
        }
    }

    for (int i = 0; i < 256; i++) {
        float value = ((h_cdf[i] - min_cdf) * 255.0f) / (total_pixels - min_cdf);
        h_lookup_table[i] = static_cast<uchar>(std::min(std::max(value, 0.0f), 255.0f));
    }

    // Copia la lookup table sulla GPU (più veloce grazie alla Pinned Memory)
    hipMemcpyAsync(d_lookup_table, h_lookup_table, 256 * sizeof(uchar), hipMemcpyHostToDevice);

    // Kernel 3: Applicazione della trasformazione (applica la tabella di lookup all'immagine)
    applyTransformation<<<gridSize, blockSize>>>(d_output, d_input, d_lookup_table, width, height);

    // Sincronizza solo prima di operazioni critiche, non per ogni kernel
    hipDeviceSynchronize();

    // Registra il tempo di fine
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calcola il tempo impiegato dai kernel
    float kernel_time;
    // Si prende il tempo passato tra i due eventi start e stop
    hipEventElapsedTime(&kernel_time, start, stop); // Serve calcolare tempo totale esecuzione dei 3 kernel senza considerare copie di memoria (hipMemcpy) --> prende il tmepo tra i due eventi (start e stop)
    std::cout << "--> Tempo di esecuzione solo dei kernel CUDA: " << kernel_time << " ms" << std::endl;

    // Copia il risultato sulla CPU (più veloce grazie alla Pinned Memory)
    hipMemcpy(output.data, d_output, total_pixels * sizeof(uchar), hipMemcpyDeviceToHost); // Questa operazione è quella che rallenta l'esecuzione

    // Libera memoria GPU
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_hist);
    hipFree(d_cdf);
    hipFree(d_lookup_table);

    // Libera la memoria pinned sulla CPU
    hipHostFree(h_cdf);
    hipHostFree(h_lookup_table);

    // Distrugge gli eventi CUDA
    hipEventDestroy(start);
    hipEventDestroy(stop);
}



